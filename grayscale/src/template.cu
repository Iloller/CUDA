#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
 * example application.
 * Host code.
 */

 // includes, system
 #include <stdlib.h>
 #include <stdio.h>
 #include <string.h>
 #include <math.h>
 #include "lodepng.h"
 
 // includes CUDA
 #include "hip/hip_runtime.h"
 #include ""
 
 // includes, project
 #include <hip/hip_runtime_api.h>
 #include <helper_functions.h> // helper functions for SDK examples
 
 ////////////////////////////////////////////////////////////////////////////////
 // declaration, forward
 void runTest(int argc, char** argv, int index);
 
 #define COUNT 5
 int BLOCKSIZE = 1;
 
 extern "C" void computeGrayScale(unsigned char* r, unsigned char* g, unsigned char* b, unsigned char* out, unsigned int ARRAYSIZE);
 
 __global__ void testGrayScale(unsigned char* r, unsigned char* g, unsigned char* b, unsigned char* out, unsigned int ARRAYSIZE) {
	 int idx = blockIdx.x * blockDim.x + threadIdx.x;
	 if (idx < ARRAYSIZE) {
		 out[idx] = 0.2126 * r[idx] + 0.7152 * g[idx] + 0.0722 * b[idx];
	 }
 }
 
 float cpuResults[COUNT];
 float gpuCalcResults[COUNT];
 float gpuTotalResults[COUNT];
 
 unsigned height, width;
 unsigned char* r, * g, * b, * a, * out;
 
 ////////////////////////////////////////////////////////////////////////////////
 // Program main
 ////////////////////////////////////////////////////////////////////////////////
 void encodeOneStep(const char* filename) {
	 /*Encode the image*/
	 unsigned char* image = (unsigned char*)malloc(4 * width * height * sizeof(unsigned char));
	 for (int y = 0; y < height; y++) {
		 for (int x = 0; x < width; x++) {
			 /*get RGBA components*/
			 image[4 * y * width + 4 * x + 0] = out[y * width + x]; /*red*/
			 image[4 * y * width + 4 * x + 1] = out[y * width + x]; /*green*/
			 image[4 * y * width + 4 * x + 2] = out[y * width + x]; /*blue*/
			 image[4 * y * width + 4 * x + 3] = a[y * width + x]; /*alpha*/
		 }
	 }
 
	 unsigned error = lodepng_encode32_file(filename, image, width, height);
 
	 /*if there's an error, display it*/
	 if (error)
		 printf("error %u: %s\n", error, lodepng_error_text(error));
	 free(r);
	 free(g);
	 free(b);
	 free(a);
	 free(image);
 
 }
 
 void decodeOneStep(const char* filename) {
	 unsigned error;
	 unsigned char* image = 0;
 
	 error = lodepng_decode32_file(&image, &width, &height, filename);
	 if (error)
		 printf("error %u: %s\n", error, lodepng_error_text(error));
 
	 r = (unsigned char*)malloc(width * height * sizeof(unsigned char));
	 g = (unsigned char*)malloc(width * height * sizeof(unsigned char));
	 b = (unsigned char*)malloc(width * height * sizeof(unsigned char));
	 a = (unsigned char*)malloc(width * height * sizeof(unsigned char));
 
	 for (int y = 0; y < height; y++) {
		 for (int x = 0; x < width; x++) {
			 /*get RGBA components*/
			 r[y * width + x] = image[4 * y * width + 4 * x + 0]; /*red*/
			 g[y * width + x] = image[4 * y * width + 4 * x + 1]; /*green*/
			 b[y * width + x] = image[4 * y * width + 4 * x + 2]; /*blue*/
			 a[y * width + x] = image[4 * y * width + 4 * x + 3]; /*alpha*/
		 }
	 }
	 free(image);
 }
 
 int main(int argc, char** argv) {
	 const char* filename = argc > 1 ? argv[1] : "original.png";
 
	 decodeOneStep(filename);
 
	 out = (unsigned char*)malloc(width * height * sizeof(unsigned char));
 
	 FILE* file;
	 char* str = (char*)malloc(50);
	 snprintf(str, 50, "opgave-3-t.csv");
	 file = fopen(str, "w");
	 free(str);
 
	 fprintf(file, "BLOCKSIZE;CPU_TIME;GPU_CALCULATION;GPU_TOTAL\n");
	 while (BLOCKSIZE <= 1024) {
		 float cpuAvg = 0;
		 float gpuCalcAvg = 0;
		 float gpuTotalAvg = 0;
		 for (int i = 0; i < COUNT; i++) {
			 runTest(argc, argv, i);
			 cpuAvg += cpuResults[i];
			 gpuCalcAvg += gpuCalcResults[i];
			 gpuTotalAvg += gpuTotalResults[i];
		 }
 
		 cpuAvg = cpuAvg / COUNT;
		 gpuCalcAvg = gpuCalcAvg / COUNT;
		 gpuTotalAvg = gpuTotalAvg / COUNT;
		 fprintf(file, "%i;%f;%f;%f\n", BLOCKSIZE, cpuAvg, gpuCalcAvg, gpuTotalAvg);
		 printf("%i;%f;%f;%f\n", BLOCKSIZE, cpuAvg, gpuCalcAvg, gpuTotalAvg);
		 BLOCKSIZE++;
	 }
 
	 encodeOneStep("grayscale-t.png");
 
	 free(out);
	 fclose(file);
 
	 return 0;
 }
 
 ////////////////////////////////////////////////////////////////////////////////
 //! Run a simple test for CUDA
 ////////////////////////////////////////////////////////////////////////////////
 void runTest(int argc, char** argv, int index) { 
	 unsigned int ARRAYSIZE = height * width;
	 unsigned int nBlocks = ARRAYSIZE / BLOCKSIZE + (ARRAYSIZE % BLOCKSIZE == 0 ? 0 : 1);
 
	 //declare variables
	 unsigned char* out_host;
	 unsigned char* r_dev, * g_dev, * b_dev, * out_dev;
	 //allocate arrays on host
	 out_host = (unsigned char*)malloc(ARRAYSIZE * sizeof(unsigned char));
 
	 for (unsigned int i = 0; i < ARRAYSIZE; i++) {
		 out_host[i] = 0;
	 }
 
	 StopWatchInterface* timerCPU = 0;
	 hipEvent_t startTotal, startCalc, stopTotal, stopCalc;
	 float timeTotal = 0;
	 float timeCalc = 0;
	 hipEventCreate(&startTotal);
	 hipEventCreate(&stopTotal);
	 hipEventCreate(&startCalc);
	 hipEventCreate(&stopCalc);
	 sdkCreateTimer(&timerCPU);
 
	 //printf("Starting GPU...\n");
	 hipEventRecord(startTotal);
 
	 //allocate arrays on device
	 hipMalloc((void**)&r_dev, ARRAYSIZE * sizeof(unsigned char));
	 hipMalloc((void**)&g_dev, ARRAYSIZE * sizeof(unsigned char));
	 hipMalloc((void**)&b_dev, ARRAYSIZE * sizeof(unsigned char));
	 hipMalloc((void**)&out_dev, ARRAYSIZE * sizeof(unsigned char));
 
	 //Step 1: Copy data to GPU memory
 
	 hipMemcpy(r_dev, r, ARRAYSIZE * sizeof(unsigned char),
		 hipMemcpyHostToDevice);
	 hipMemcpy(g_dev, g, ARRAYSIZE * sizeof(unsigned char),
		 hipMemcpyHostToDevice);
	 hipMemcpy(b_dev, b, ARRAYSIZE * sizeof(unsigned char),
		 hipMemcpyHostToDevice);
	 hipMemcpy(out_dev, out_host, ARRAYSIZE * sizeof(unsigned char),
		 hipMemcpyHostToDevice);
 
	 //Step 2 & 3: RUN
	 hipEventRecord(startCalc);
	 testGrayScale << < nBlocks, BLOCKSIZE >> > (r_dev, g_dev, b_dev, out_dev, ARRAYSIZE);
	 hipEventRecord(stopCalc);
	 hipEventSynchronize(stopCalc);
 
 
 
	 // check if kernel execution generated and error
	 getLastCudaError("Kernel execution failed");
 
	 //Step 4: Retrieve result
	 hipMemcpy(out_host, out_dev, ARRAYSIZE * sizeof(unsigned char),
		 hipMemcpyDeviceToHost);
 
	 hipEventRecord(stopTotal);
	 hipEventSynchronize(stopTotal);
	 hipEventElapsedTime(&timeCalc, startCalc, stopCalc);
	 hipEventElapsedTime(&timeTotal, startTotal, stopTotal);
 
	 //printf("Starting CPU...\n");
 
	 sdkStartTimer(&timerCPU);
 
	 computeGrayScale(r, g, b, out_host, ARRAYSIZE);
 
	 sdkStopTimer(&timerCPU);
 
	 for (unsigned int i = 0; i < ARRAYSIZE; i++) {
		 out[i] = out_host[i];
	 }
 
	 // RESULTS
	 cpuResults[index] = sdkGetTimerValue(&timerCPU);
	 gpuCalcResults[index] = timeCalc;
	 gpuTotalResults[index] = timeTotal;
 
	 //rest of program (Other 4 steps go here)
	 //end of  program
	 //cleanup: VERY IMPORTANT!!!
	 sdkDeleteTimer(&timerCPU);
	 free(out_host);
	 hipFree(r_dev);
	 hipFree(g_dev);
	 hipFree(b_dev);
	 hipFree(out_dev);
 }
 