#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
 * example application.
 * Host code.
 */

 // includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "lodepng.h"

// includes CUDA
#include "hip/hip_runtime.h"
#include ""

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv, int index);

#define COUNT 10
int BLOCKSIZE = 1;

extern "C" void computeGrayScale(unsigned char* r, unsigned char* g, unsigned char* b, unsigned int ARRAYSIZE);

extern "C" void computeEdges(unsigned char* in, unsigned char* out, unsigned int width, unsigned int height);

__global__ void testEdges(unsigned char* in, unsigned char* out, unsigned int width, unsigned int height) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (0 < idx && idx < width - 1 && 0 < idy && idy < height - 1) {
		float tmpVert = -1 * in[(idy - 1) * width + idx - 1] + 0 * in[(idy - 1) * width + idx] + 1 * in[(idy - 1) * width + idx + 1];
		tmpVert += -2 * in[idy * width + idx - 1] + 0 * in[idy * width + idx] + 2 * in[idy * width + idx + 1];
		tmpVert += -1 * in[(idy + 1) * width + idx - 1] + 0 * in[(idy + 1) * width + idx] + 1 * in[(idy + 1) * width + idx + 1];

		float tmpHor = 1 * in[(idy - 1) * width + idx - 1] + 2 * in[(idy - 1) * width + idx] + 1 * in[(idy - 1) * width + idx + 1];
		tmpHor += 0 * in[idy * width + idx - 1] + 0 * in[idy * width + idx] + 0 * in[idy * width + idx + 1];
		tmpHor += -1 * in[(idy + 1) * width + idx - 1] + -2 * in[(idy + 1) * width + idx] + -1 * in[(idy + 1) * width + idx + 1];

		float mag = sqrt(tmpVert * tmpVert + tmpHor * tmpHor);

		if (mag > 255)
		{
			mag = 255;
		}
		out[idy * width + idx] = mag;
	}
}

float cpuResults[COUNT];
float gpuCalcResults[COUNT];
float gpuTotalResults[COUNT];

unsigned height, width;
unsigned char* r, * g, * b, * a, * out;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
void encodeOneStep(const char* filename) {
	/*Encode the image*/
	unsigned char* image = (unsigned char*)malloc(4 * width * height * sizeof(unsigned char));
	for (int y = 0; y < height; y++) {
		for (int x = 0; x < width; x++) {
			/*get RGBA components*/
			image[4 * y * width + 4 * x + 0] = out[y * width + x]; /*red*/
			image[4 * y * width + 4 * x + 1] = out[y * width + x]; /*green*/
			image[4 * y * width + 4 * x + 2] = out[y * width + x]; /*blue*/
			image[4 * y * width + 4 * x + 3] = a[y * width + x]; /*alpha*/
		}
	}

	unsigned error = lodepng_encode32_file(filename, image, width, height);

	/*if there's an error, display it*/
	if (error)
		printf("error %u: %s\n", error, lodepng_error_text(error));
	free(r);
	free(g);
	free(b);
	free(a);
	free(image);

}

void decodeOneStep(const char* filename) {
	unsigned error;
	unsigned char* image = 0;

	error = lodepng_decode32_file(&image, &width, &height, filename);
	if (error)
		printf("error %u: %s\n", error, lodepng_error_text(error));

	r = (unsigned char*)malloc(width * height * sizeof(unsigned char));
	g = (unsigned char*)malloc(width * height * sizeof(unsigned char));
	b = (unsigned char*)malloc(width * height * sizeof(unsigned char));
	a = (unsigned char*)malloc(width * height * sizeof(unsigned char));

	for (int y = 0; y < height; y++) {
		for (int x = 0; x < width; x++) {
			/*get RGBA components*/
			r[y * width + x] = image[4 * y * width + 4 * x + 0]; /*red*/
			g[y * width + x] = image[4 * y * width + 4 * x + 1]; /*green*/
			b[y * width + x] = image[4 * y * width + 4 * x + 2]; /*blue*/
			a[y * width + x] = image[4 * y * width + 4 * x + 3]; /*alpha*/
		}
	}
	free(image);
}

int main(int argc, char** argv) {
	const char* filename = argc > 1 ? argv[1] : "image.png";

	decodeOneStep(filename);

	computeGrayScale(r, g, b, width * height);

	out = (unsigned char*)malloc(width * height * sizeof(unsigned char));

	FILE* file;
	char* str = (char*)malloc(50);
	snprintf(str, 50, "opgave-4.csv");
	file = fopen(str, "w");
	free(str);

	fprintf(file, "BLOCKSIZE;CPU_TIME;GPU_CALCULATION;GPU_TOTAL\n");
	while (BLOCKSIZE <= 1024) {
		float cpuAvg = 0;
		float gpuCalcAvg = 0;
		float gpuTotalAvg = 0;
		for (int i = 0; i < COUNT; i++) {
			runTest(argc, argv, i);
			cpuAvg += cpuResults[i];
			gpuCalcAvg += gpuCalcResults[i];
			gpuTotalAvg += gpuTotalResults[i];
		}

		cpuAvg = cpuAvg / COUNT;
		gpuCalcAvg = gpuCalcAvg / COUNT;
		gpuTotalAvg = gpuTotalAvg / COUNT;
		fprintf(file, "%i;%f;%f;%f\n", BLOCKSIZE, cpuAvg, gpuCalcAvg, gpuTotalAvg);
		printf("%i;%f;%f;%f\n", BLOCKSIZE, cpuAvg, gpuCalcAvg, gpuTotalAvg);
		BLOCKSIZE++;
	}

	encodeOneStep("edges.png");
	free(out);
	fclose(file);

	return 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char** argv, int index) {

	int nBlocksHor = width / BLOCKSIZE + (width % BLOCKSIZE == 0 ? 0 : 1);
	int nBlocksver = height;


	// setup execution parameters
	dim3 grid(nBlocksHor, nBlocksver, 1);

	unsigned int ARRAYSIZE = height * width;

	//declare variables
	unsigned char* out_host;
	unsigned char* in_dev, * out_dev;
	//allocate arrays on host
	out_host = (unsigned char*)malloc(ARRAYSIZE * sizeof(unsigned char));

	for (unsigned int i = 0; i < ARRAYSIZE; i++) {
		out_host[i] = 0;
	}

	StopWatchInterface* timerCPU = 0;
	hipEvent_t startTotal, startCalc, stopTotal, stopCalc;
	float timeTotal = 0;
	float timeCalc = 0;
	hipEventCreate(&startTotal);
	hipEventCreate(&stopTotal);
	hipEventCreate(&startCalc);
	hipEventCreate(&stopCalc);
	sdkCreateTimer(&timerCPU);

	//printf("Starting GPU...\n");
	hipEventRecord(startTotal);

	//allocate arrays on device
	hipMalloc((void**)&in_dev, ARRAYSIZE * sizeof(unsigned char));
	hipMalloc((void**)&out_dev, ARRAYSIZE * sizeof(unsigned char));

	//Step 1: Copy data to GPU memory
	hipMemcpy(in_dev, r, ARRAYSIZE * sizeof(unsigned char),
		hipMemcpyHostToDevice);
	hipMemcpy(out_dev, out_host, ARRAYSIZE * sizeof(unsigned char),
		hipMemcpyHostToDevice);

	//Step 2 & 3: RUN
	hipEventRecord(startCalc);
	testEdges << < grid, BLOCKSIZE >> > (in_dev, out_dev, width, height);
	hipEventRecord(stopCalc);
	hipEventSynchronize(stopCalc);

	// check if kernel execution generated and error
	getLastCudaError("Kernel execution failed");

	//Step 4: Retrieve result
	hipMemcpy(out_host, out_dev, ARRAYSIZE * sizeof(unsigned char),
		hipMemcpyDeviceToHost);

	hipEventRecord(stopTotal);
	hipEventSynchronize(stopTotal);
	hipEventElapsedTime(&timeCalc, startCalc, stopCalc);
	hipEventElapsedTime(&timeTotal, startTotal, stopTotal);

	for (unsigned int i = 0; i < ARRAYSIZE; i++) {
		out[i] = out_host[i];
	}

	//printf("Starting CPU...\n");

	sdkStartTimer(&timerCPU);

	computeEdges(r, out_host, width, height);

	sdkStopTimer(&timerCPU);

	// RESULTS
	cpuResults[index] = sdkGetTimerValue(&timerCPU);
	gpuCalcResults[index] = timeCalc;
	gpuTotalResults[index] = timeTotal;

	//rest of program (Other 4 steps go here)
	//end of  program
	//cleanup: VERY IMPORTANT!!!
	sdkDeleteTimer(&timerCPU);
	free(out_host);
	hipFree(in_dev);
	hipFree(out_dev);
}
